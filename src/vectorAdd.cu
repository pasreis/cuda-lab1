#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <iostream>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>
#include <sys/time.h>
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int
main(int argc, char** argv)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    unsigned long long numElements = 2<<20	; // <------- size!
    int threadsPerBlock = 256, option;

    while ((option = getopt(argc, argv, "n:t:")) != -1) {
    	switch (option) {
    		case 'n':
    			numElements = (unsigned long long) atoi(optarg);
    			break;
    		case 't':
    			threadsPerBlock = (int) atoi(optarg);
    			break;
    		default:
    			printf("Error!\n");
    			exit(0);
    	}
    }

    size_t size = numElements * sizeof(float);

	hipDeviceProp_t prop;
	int numDevices = 0;

	err = hipGetDeviceCount(&numDevices);

	if (err != hipSuccess) {
		fprintf(stderr, "Failed to query the number of devices!\n");
		exit(EXIT_FAILURE);
	}

	int totalMem = 0;

	for (int i = 0; i < numDevices; i++) {
		err = hipGetDeviceProperties(&prop, i);

		if (err != hipSuccess) {
			fprintf(stderr, "Failed to query the device properties!\n");
			exit(EXIT_FAILURE);
		}

		totalMem += prop.totalGlobalMem;
	}

	if (size > totalMem){
		printf("Memory exceeded!\n");
		exit(EXIT_FAILURE);
	}

    printf("[Vector addition of %d elements]\n", numElements);

    float* A, *B, *C;
    err = hipMallocManaged(&A, size);
    if (err != hipSuccess) {
    	fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
    }

    err = hipMallocManaged(&B, size);
    if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

    err = hipMallocManaged(&C, size);
    if (err != hipSuccess) {
		fprintf(stderr, "Failed to allocate host vectors!\n");
		exit(EXIT_FAILURE);
	}

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        A[i] = rand()/(float)RAND_MAX;
        B[i] = rand()/(float)RAND_MAX;
    }

	int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, numElements);
    hipDeviceSynchronize();
    err = hipGetLastError();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(A[i] + B[i] - C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(A);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(B);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(C);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("Done\n");
    return 0;
}

